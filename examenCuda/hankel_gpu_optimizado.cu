
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
#define BLOCKSIZE 4

void Print_matrix(int C[], int n)
{
   int i, j;

   for (i = 0; i < n; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d\t", C[i + j * n]);
      printf("\n");
   }
} /* Print_matrix */

void comprobar_cpu(int *A, int *sal)
{
   int i, j, res = 1;
   for (j = 1; j < N; j++)
      for (i = 0; i < N - 1; i++)
         if (A[i + j * N] != A[i + 1 + (j - 1) * N])
            res = 0;

   *sal = res;
}

__global__ void comprobar_gpu_optimizado(int *A, bool *sal)
{
   __shared__ bool cache[BLOCKSIZE];
   bool temp_result = true;
   int j = blockIdx.x + 1;
   int tid = threadIdx.x;

   while ((tid < N - 1) && temp_result)
   {
      if (A[tid + j * N] != A[tid + 1 + (j - 1) * N])
         temp_result = false;
      tid += BLOCKSIZE;
   }
   cache[threadIdx.x] = temp_result;

   __syncthreads();
   j = BLOCKSIZE/2;

   while(j!=0)
   {
      if(threadIdx.x < j)
         cache[threadIdx.x] &= cache[threadIdx.x+j];
      __syncthreads();
      j /= 2;
   }
   if(threadIdx.x == 0)
      sal[blockIdx.x] = cache[threadIdx.x];
}

int main()
{
   int i, j;
   int *A = (int *)malloc(N * N * sizeof(int));
   int *sal = (int *)malloc(N * sizeof(int));
   int salcpu;

   // rellenar matriz de numeros en CPU
   for (j = 0; j < N; j++)
      for (i = 0; i < N; i++)
      {
         A[i + N * j] = i + j - 1;
      }
   // A[3+N*4]=77;
   Print_matrix(A, N);
   comprobar_cpu(A, &salcpu);

   if (salcpu == 1)
      printf(" \n La matriz es hankel (cpu)\n");
   else
      printf(" \n La matriz no es hankel (cpu)\n");

   // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
   //  y lo que sea necesario
   // Comienzo parte GPU
   int *dev_A;
   bool *dev_sal, *sal_gpu = (bool *)malloc(N - 1 * sizeof(bool));

   bool es_hankel = true;

   hipMalloc((void **)&dev_A, N * N * sizeof(int));
   hipMalloc((void **)&dev_sal, N - 1 * sizeof(bool));

   hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);

   comprobar_gpu_optimizado<<<N - 1, BLOCKSIZE>>>(dev_A, dev_sal);

   hipMemcpy(sal_gpu, dev_sal, N - 1 * sizeof(bool), hipMemcpyDeviceToHost);

   for (i = 0; i < N - 1; i++)
      if (!sal_gpu[i])
         es_hankel = false;

   if (es_hankel)
      printf(" \n La matriz es hankel (gpu)\n");
   else
      printf(" \n La matriz no es hankel (gpu)\n");
}
