
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
#define BLOCKSIZE (N - 1)

void Print_matrix(int C[], int n)
{
   int i, j;

   for (i = 0; i < n; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d\t", C[i + j * n]);
      printf("\n");
   }
} /* Print_matrix */

void comprobar_cpu(int *A, int *sal)
{
   int i, j, res = 1;
   for (j = 1; j < N; j++)
      for (i = 0; i < N - 1; i++)
         if (A[i + j * N] != A[i + 1 + (j - 1) * N])
            res = 0;

   *sal = res;
}

__global__ void comprobar_gpu(int *A, int *sal)
{
   int i, j, res = 1;
   j = threadIdx.x + 1;
   for (i = 0; i < N - 1; i++)
      if (A[i + j * N] != A[i + 1 + (j - 1) * N])
         res = 0;

   sal[threadIdx.x] = res;
}

int main()
{
   int i, j;
   int *A = (int *)malloc(N * N * sizeof(int));
   int *sal = (int *)malloc(N * sizeof(int));
   int salcpu;

   // rellenar matriz de numeros en CPU
   for (j = 0; j < N; j++)
      for (i = 0; i < N; i++)
      {
         A[i + N * j] = i + j - 1;
      }
   A[3+N*4]=77;
   Print_matrix(A, N);
   comprobar_cpu(A, &salcpu);

   if (salcpu == 1)
      printf(" \n La matriz es hankel (cpu)\n");
   else
      printf(" \n La matriz no es hankel (cpu)\n");

   // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
   //  y lo que sea necesario
   // Comienzo parte GPU
   int *dev_A, *dev_sal;
   int *sal_gpu = (int *)malloc(BLOCKSIZE * sizeof(int));

   bool es_hankel = true;

   hipMalloc((void **)&dev_A, N * N * sizeof(int));
   hipMalloc((void **)&dev_sal, BLOCKSIZE * sizeof(int));

   hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);

   comprobar_gpu<<<1, BLOCKSIZE>>>(dev_A, dev_sal);

   hipMemcpy(sal_gpu, dev_sal, BLOCKSIZE * sizeof(int), hipMemcpyDeviceToHost);

   for (i = 0; i < BLOCKSIZE; i++)
      if (sal_gpu[i] == 0)
         es_hankel = false;

   if (es_hankel)
      printf(" \n La matriz es hankel (gpu)\n");
   else
      printf(" \n La matriz no es hankel (gpu)\n");
}
