
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 10000
#define N 10000

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}
 
__global__ void mediasmatrizcpu(double *A,  double *sal)
{
	int i,j; 
  double suma;

	j = threadIdx.x;
	suma=0;
	for(i = 0; i < M; i++)
		 suma = suma + A[i+j*M];
	sal[j] = suma / double(M);
}

int main() {
	int i,j;
	double *dev_a, *dev_c;
 
  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );

	hipMalloc((void **) &dev_a, N*M*sizeof(double) );
	hipMalloc((void **) &dev_c, N*sizeof(double) );
 
	//rellenar matriz en CPU
  for (j=0;j<N;j++)
		for(i=0;i<M;i++)
		{
			A[i+M*j]=i+j ;
    }

	//Print_matrix(A,M,N);

	//enviar vectores a GPU
	hipMemcpy( dev_a, A, N*M*sizeof(double) , hipMemcpyHostToDevice );
	hipMemcpy( dev_c, sal1, N*sizeof(double) , hipMemcpyHostToDevice );

  mediasmatrizcpu<<<1,N>>>(dev_a,dev_c);

	//obtener el resultado de vuelta en la CPU
	hipMemcpy( sal1, dev_c, N*sizeof(double), hipMemcpyDeviceToHost );

	for (j=0;j<N;j++)
		printf("media columna %d = %f  \n",j,sal1[j]);

  free(A);
  free(sal1);
}
