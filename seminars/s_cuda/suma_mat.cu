
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 100
#define N 100

void printMatrix (int *c)
{
	int i,j;
	for (i=0;i<N;i++)
	{
		for(j=0;j<M;j++)
		{
			printf("%d\t", c[i+M*j]);
		}
		printf("\n");
	}
	printf("\n");
}

__global__ void add(int *a, int *b, int *c)
{
	int tid_x =  threadIdx.x+blockIdx.x*blockDim.x;
	int tid_y =  threadIdx.y+blockIdx.y*blockDim.y;
	int index = tid_x+tid_y*M;
	if(index<N*M)
		c[index]=a[index]+b[index];
}
 
int main() {
	int i,j;
	int a[N*M], b[N*M], c[N*M];
	int *dev_a, *dev_b, *dev_c;

	//reservar memoria en GPU
	hipMalloc((void **) &dev_a, M*N*sizeof(int) );
	hipMalloc((void **) &dev_b, M*N*sizeof(int) );
	hipMalloc((void **) &dev_c, M*N*sizeof(int) );

	//rellenar vectores en CPU
	for (i=0;i<N;i++)
	{
		for(j=0;j<M;j++)
		{
			a[i+M*j]=-i+12*j;
			b[i+M*j]=2*i-j;
		}
	}
	printf("Matrix A:\n");
	printMatrix(a);
	printf("Matrix B:\n");
	printMatrix(b);

	//enviar vectores a GPU
	hipMemcpy( dev_a, a, M*N*sizeof(int) , hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, M*N*sizeof(int) , hipMemcpyHostToDevice );
	hipMemcpy( dev_c, c, M*N*sizeof(int) , hipMemcpyHostToDevice );

	dim3 g(2,3);
	dim3 bl(4,4);
	//llamar al Kernel
	add<<<g,bl>>>(dev_a,dev_b,dev_c);

	//obtener el resultado de vuelta en la CPU
	hipMemcpy( c, dev_c, M*N*sizeof(int), hipMemcpyDeviceToHost );

	printf("Matrix result: \n");
	printMatrix(c);
		
	hipFree(dev_a) ;
	hipFree(dev_b) ;
	hipFree(dev_c) ;
}
	
	
