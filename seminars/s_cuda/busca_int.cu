
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
#define BLOCKSIZE 4
void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N;j++)
       for(i=0;i<N-1;i++)
            if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
              cant++;

 *sal=cant;
}

__global__ void contar_cuda(int *A, int *sal, int num1, int num2)
{
    int tid = threadIdx.x;
    sal[tid] = 0;
    
    for (int i=0;i<N-1;i++)
        if((A[i+tid*N] == num1) && (A[i+1+tid*N] == num2))
            sal[tid]++;
}

__global__ void contar_cuda_2(int *A, int *sal, int num1, int num2)
{
    __shared__ int cache[BLOCKSIZE];

    int i;
    int tid_col = blockIdx.x;
    int tid = threadIdx.x;

    sal[tid_col] = 0;

    for (i = tid ; i < N-1; i += blockDim.x)
    {
        if((A[i+tid_col*N] == num1) && (A[i+1+tid_col*N] == num2))
            cache[tid]++;
    }
    if (tid == 0)
        for(i = 0; i < blockDim.x; i++)
            sal[tid_col] += cache[i];
}
 
int main() {
  int i,j;
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;

 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
      A[i+N*j]=rand()% 10;
     
  Print_matrix(A,N);
  contar_int(A,&salcpu,6,3);
  printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu
  int *dev_A;
  int *dev_sal;
  int sal_gpu = 0;

  hipMalloc((void **) &dev_A, N*N*sizeof(int) );
  hipMalloc((void **) &dev_sal, N*sizeof(int) );
  hipMemcpy(dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);

//   contar_cuda<<<1, N>>>(dev_A, dev_sal, 6,3);
  contar_cuda_2<<<N, BLOCKSIZE>>>(dev_A, dev_sal, 6,3);

  hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );

  for(i = 0; i < N; i++)
    sal_gpu += sal[i] ;
  printf(" \n En gpu se cuentan %d secuencias %d %d \n",sal_gpu, 6,3);

  free(A);
}