
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 8
#define N 8

#define W (M-2)
#define H (N-2)

void printMatrix (double *c, int m, int n)
{
	int i,j;
	for (i=0;i<n;i++)
	{
		for(j=0;j<m;j++)
		{
			printf("%f ", c[i+m*j]);
		}
		printf("\n");
	}
	printf("\n");
}

__global__ void filter(double *a, double *c)
{
	int tid_x =  threadIdx.x+blockIdx.x*blockDim.x;
	int tid_y =  threadIdx.y+blockIdx.y*blockDim.y;

	int tp_x = tid_x+1;
	int tp_y = tid_y+1;

	int index = tid_x+tid_y*W;
	if(index<W*H)
		c[index]=( a[(tp_x-1)+tp_y*W] + a[(tp_x+1)+tp_y*W] +
							 a[tp_x+(tp_y-1)*W] + a[tp_x+(tp_y+1)*W] +
							 a[tp_x+tp_y*W] ) / 5.0;
	printf("c[%d]=%f\n", index,c[index]);
}
 
int main() {
	int i,j;

	double a[N*M], c[W*H];
	double *dev_a, *dev_c;

	//reservar memoria en GPU
	hipMalloc((void **) &dev_a, M*N*sizeof(double) );
	hipMalloc((void **) &dev_c, W*H*sizeof(double) );

	//rellenar vectores en CPU
	for (i=0;i<N;i++)
	{
		for(j=0;j<M;j++)
		{
			a[i+M*j]=-i+12*j;
		}
	}
	printf("Matrix Inp:\n");
	printMatrix(a, M, N);

	//enviar vectores a GPU
	hipMemcpy( dev_a, a, M*N*sizeof(double) , hipMemcpyHostToDevice );

	dim3 g(2,2);
	dim3 bl(4,4);
	//llamar al Kernel
	filter<<<g,bl>>>(dev_a, dev_c);

	//obtener el resultado de vuelta en la CPU
	hipMemcpy( c, dev_c, W*H*sizeof(double), hipMemcpyDeviceToHost );

	printf("Matrix result: \n");
	printMatrix(c, W, H);
		
	hipFree(dev_a) ;
	hipFree(dev_c) ;
}
	
	
