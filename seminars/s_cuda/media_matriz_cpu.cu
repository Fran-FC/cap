
#include <hip/hip_runtime.h>
#include <stdio.h>


#define M 5
#define N 6


void mediasmatrizcpu(double *A,  double *sal)
{
  int i,j; 
  double suma;
  for(j=0;j<N;j++)
    {suma=0;
    for(i=0;i<M;i++)
       suma=suma+A[i+j*M];
    sal[j]=suma/double(M);
    }
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */
 
 int main() {
  int i,j;

 
  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );

 
 //rellenar matriz en CPU
  for (j=0;j<N;j++)
    for(i=0;i<M;i++)
   {
      A[i+M*j]=i+j ;
    }

  Print_matrix(A,M,N);

  mediasmatrizcpu(A,sal1);

   for (j=0;j<N;j++)
    printf("media columna %d = %f  \n",j,sal1[j]);

  free(A);
  free(sal1);

  }
	
	
