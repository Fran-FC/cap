
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

void media(double *a,  double *c)
{
  int i;
  for (i=0;i<N-2;i++)
		c[i]=(a[i]+a[i+1]+a[i+2])/3.0f;
}
 
__global__ void media_cuda(double *a, double *c)
{
	int tid	= blockIdx.x;
	c[tid] = (a[tid]+a[tid+1]+a[tid+2]) / 3.0f;
//	while(tid<N)
//	{
//		c[tid] = (a[tid]+a[tid+1]+a[tid+2]) / 3.0f;
//		tid += 1;
//	}
}

int main() {
	double a[N], c[N];
	double *dev_a, *dev_c;
	int i;

	hipMalloc((void **) &dev_a, N*sizeof(double) );
	hipMalloc((void **) &dev_c, N*sizeof(double) );

	//rellenar vectores en CPU
	for (i=0;i<N;i++)
	{
			a[i]=i*i;
			//a[i]=i;
	}

	//enviar vectores a GPU
	hipMemcpy( dev_a, a, N*sizeof(double) , hipMemcpyHostToDevice );
	hipMemcpy( dev_c, c, N*sizeof(double) , hipMemcpyHostToDevice );

	//llamar al kernel
	int n_blocks = N-2;
	media_cuda<<<n_blocks,1>>>(dev_a, dev_c);

	//obtener el resultado de vuelta en la CPU
	hipMemcpy( c, dev_c, N*sizeof(double), hipMemcpyDeviceToHost );

	for (i=0;i<N-2;i++)
		printf("  %f\n",  c[i]);

	hipFree(dev_a) ;
	hipFree(dev_c) ;
}
	
	
